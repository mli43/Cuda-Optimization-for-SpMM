#include "spmm_cusparse.hpp"
#include "format.hpp"

namespace cuspmm {

template <typename DataT>
DenseMatrix<DataT>* cusparseTest(SparseMatrix<DataT>* a, DenseMatrix<DataT>* b) {
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    DenseMatrix<DataT>* c = new DenseMatrix<DataT>(a->numRows, b->numCols, true, ORDERING::ROW_MAJOR);

    if (b->ordering != ORDERING::COL_MAJOR) {
        b->toOrdering(ORDERING::COL_MAJOR);
    }

    auto t1 = std::chrono::high_resolution_clock::now();

    CHECK_CUSPARSE(hipsparseCreate(&handle));
    a->setCusparseSpMatDesc(&matA);
    b->setCusparseDnMatDesc(&matB);
    c->setCusparseDnMatDesc(&matC);

    float alpha = 1.0f, beta = 0.f;
    void* dBuffer = nullptr;
    size_t buffersize = 0;
    hipsparseSpMMAlg_t alg = a->getCusparseAlg();
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, alg, &buffersize));
    cudaCheckError(hipMalloc(&dBuffer, buffersize));
    auto t2 = std::chrono::high_resolution_clock::now();
    CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, alg, dBuffer));
    hipDeviceSynchronize();
    auto t3 = std::chrono::high_resolution_clock::now();

    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    cudaCheckError(hipFree(dBuffer));
    auto t4 = std::chrono::high_resolution_clock::now();

    auto prepTime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(t3 - t2);
    auto epilogueTime = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3);

    std::cout << "cusparse prep time (us):" << prepTime.count() << ','
              << "cusparse kernel time (us):" << kernelTime.count() << ','
              << "cusparse epilogue time (us):" << epilogueTime.count() << std::endl;
    
    return c;
}

template DenseMatrix<float>* cusparseTest(SparseMatrix<float>* a, DenseMatrix<float>* b);

}

