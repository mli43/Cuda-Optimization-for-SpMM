#include "formats/dense.hpp"

namespace cuspmm {
template <typename T>
DenseMatrix<T>::DenseMatrix(std::string filePath) {
    // Files contains only row-major contents
    this->ordering = ORDERING::ROW_MAJOR;
    this->data = nullptr;
    this->onDevice = false;

    std::ifstream inputFile(filePath);
    std::string line;

    if (!inputFile.is_open()) {
        std::cerr << "File " << filePath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    inputFile >> this->numRows >> this->numCols;
    std::getline(inputFile, line); // Discard the header line

    this->allocateSpace(this->onDevice);

    for (int i = 0; i < this->numRows; i++) {
        std::getline(inputFile, line);
        std::istringstream iss(line);
        for (int j = 0; j < this->numCols; j++) {
            iss>> this->data[i * this->numCols + j];
        }
    }

}

template <typename T>
DenseMatrix<T>::DenseMatrix(Matrix::metadataType numRows,
                    Matrix::metadataType numCols, bool onDevice, ORDERING ordering) {
    this->numRows = numRows;
    this->numCols = numCols;
    this->onDevice = onDevice;
    this->ordering = ordering;
    this->data = nullptr;
    this->allocateSpace(onDevice);
}

template <typename T>
DenseMatrix<T>::DenseMatrix(DenseMatrix<T>* source, bool onDevice) {
    this->numRows = source->numRows;
    this->numCols = source->numCols;
    this->onDevice = onDevice;
    this->ordering = source->ordering;
    this->data = nullptr;
    this->allocateSpace(this->onDevice);
    this->copyData(source);
}

template <typename T>
DenseMatrix<T>::~DenseMatrix() {
    this->freeSpace();
}

template <typename T>
bool DenseMatrix<T>::copyData(DenseMatrix<T>* source) {
    this->assertSameShape(source);
    hipMemcpyKind type;
    if (source->onDevice && this->onDevice) {
        type = hipMemcpyDeviceToDevice;
    } else if (source->onDevice && !this->onDevice) {
        type = hipMemcpyDeviceToHost;
    } else if (!source->onDevice && this->onDevice) {
        type = hipMemcpyHostToDevice;
    } else {
        type = hipMemcpyHostToHost;
    }

    cudaCheckError(
        hipMemcpy(this->data, source->data,
                    (this->numRows * this->numCols) * sizeof(T),
                    type));
    
    return true;
}

template <typename T>
void DenseMatrix<T>::assertSameShape(DenseMatrix<T>* target) {
    assert(
        this->numRows == target->numRows &&
        this->numCols == target->numCols
    );
}

template <typename T>
DenseMatrix<T>* DenseMatrix<T>::copy2Device() {
    assert(this->onDevice == false);
    assert(this->data != nullptr);

    DenseMatrix<T>* newMatrix = new DenseMatrix<T>(this, true);
    return newMatrix;
}

template <typename T>
DenseMatrix<T>* DenseMatrix<T>::copy2Host() {
    assert(this->onDevice == true);
    assert(this->data != nullptr);

    DenseMatrix<T>* newMatrix = new DenseMatrix<T>(this, false);
    return newMatrix;
}

template <typename T>
bool DenseMatrix<T>::toOrdering(ORDERING newOrdering) {
    if (this->ordering == newOrdering) {
        return true;
    }

    // Malloc new space
    size_t totalSize = this->numRows * this->numCols * sizeof(T);
    T* newData;
    cudaCheckError(hipHostMalloc(&newData, totalSize));

    // If on device, copy to host
    if (this->onDevice) {
        cudaCheckError(hipMemcpy(newData, this->data, totalSize, hipMemcpyDeviceToHost));
    }

    if (this->ordering == ORDERING::ROW_MAJOR && newOrdering == ORDERING::COL_MAJOR) {
        // Reorganize
        for (int r = 0; r < this->numRows; r++) {
            for (int c = 0; c < this->numCols; c++) {
                newData[ColMjIdx(r, c, this->numRows)] = this->data[RowMjIdx(r, c, this->numCols)];
            }
        }
    } else if (this->ordering == ORDERING::COL_MAJOR && newOrdering == ORDERING::ROW_MAJOR) {
        // Reorganize
        for (int r = 0; r < this->numRows; r++) {
            for (int c = 0; c < this->numCols; c++) {
                newData[RowMjIdx(r, c, this->numCols)] = this->data[ColMjIdx(r, c, this->numRows)];
            }
        }
    } else {
        throw std::runtime_error("Incorrect ordering value");
        return false;
    }

    this->freeSpace();
    this->allocateSpace(this->onDevice);
    if (this->onDevice) {
        cudaCheckError(hipMemcpy(this->data, newData, totalSize, hipMemcpyHostToDevice));
    } else {
        this->data = newData;
    }

    this->ordering = newOrdering;

    return true;
}

template <typename T>
bool DenseMatrix<T>::save2File(std::string filePath) {
    using mt = Matrix::metadataType;
    assert(!this->onDevice);
    assert(this->ordering == ORDERING::ROW_MAJOR);

    std::ofstream outputFile(filePath);
    if (!outputFile.is_open()) {
        std::cerr << "Cannot open output file " << filePath << std::endl;
        return false;
    }

    outputFile << this->numRows << ' ' << this->numCols << std::endl;
    for (mt r = 0; r < this->numRows; r++) {
        for (mt c = 0; c < this->numCols; c++) {
            outputFile << this->data[r * this->numCols + c] << ' ';
        }
        outputFile << std::endl;
    }

    return true;
}

template <typename T>
bool DenseMatrix<T>::allocateSpace(bool onDevice) {
    assert(this->data == nullptr);

    size_t totalSize = this->numRows * this->numCols * sizeof(T);
    if (onDevice) {
        cudaCheckError(hipMalloc(
            &this->data, totalSize));
        cudaCheckError(hipMemset(this->data, 0, totalSize));
    } else {
        cudaCheckError(hipHostMalloc(
            &this->data, totalSize));
        std::memset(this->data, 0, totalSize);
    }
    this->onDevice = onDevice;

    return true;
}

template <typename T>
bool DenseMatrix<T>::freeSpace() {
    if (this->onDevice) {
        cudaCheckError(hipFree(this->data));
    } else {
        cudaCheckError(hipHostFree(this->data));
    }
    this->data = nullptr;
    return true;
}

template class DenseMatrix<float>;
template class DenseMatrix<double>;
}