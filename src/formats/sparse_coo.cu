#include "hip/hip_runtime.h"
#include "commons.hpp"
#include "formats/sparse_coo.hpp"

namespace cuspmm {

template <typename DT, typename MT>
SparseMatrixCOO<DT, MT>::SparseMatrixCOO() : SparseMatrix<DT, MT>() {
    this->rowIdxs = nullptr;
    this->colIdxs = nullptr;
}

template <typename DT, typename MT>
SparseMatrixCOO<DT, MT>::SparseMatrixCOO(std::string filePath) {
    this->rowIdxs = nullptr;
    this->colIdxs = nullptr;
    this->onDevice = false;

    std::ifstream inputFile(filePath);
    std::string line;

    if (!inputFile.is_open()) {
        std::cerr << "File " << filePath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    inputFile >> this->numRows >> this->numCols >> this->numNonZero;
    constexpr auto max_size = std::numeric_limits<std::streamsize>::max();
    inputFile.ignore(max_size, '\n');

    this->allocateSpace(false);

    // Read row ptrs
    for (size_t i = 0; i < this->numNonZero; i++) {
        inputFile >> this->rowIdxs[i] >> this->colIdxs[i] >> this->data[i];
    }

    inputFile.close();
}

template <typename DT, typename MT>
SparseMatrixCOO<DT, MT>::SparseMatrixCOO(MT numRows,
                                    MT numCols,
                                    MT numNonZero,
                                    bool onDevice)
    : rowIdxs(nullptr), colIdxs(nullptr) {
    this->numRows = numRows;
    this->numCols = numCols;
    this->numNonZero = numNonZero;
    this->onDevice = onDevice;
    this->allocateSpace(onDevice);
}

template <typename DT, typename MT> SparseMatrixCOO<DT, MT>::~SparseMatrixCOO() {
    if (this->rowIdxs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->rowIdxs));
        } else {
            cudaCheckError(hipHostFree(this->rowIdxs));
        }
    }

    if (this->colIdxs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->colIdxs));
        } else {
            cudaCheckError(hipHostFree(this->colIdxs));
        }
    }

    if (this->data != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->data));
        } else {
            cudaCheckError(hipHostFree(this->data));
        }
    }
}

template <typename DT, typename MT>
void SparseMatrixCOO<DT, MT>::setCusparseSpMatDesc(hipsparseSpMatDescr_t *matDescP) {
    hipDataType dt;
    if constexpr (std::is_same<DT, half>::value) {
        dt = HIP_R_16F;
    } else if constexpr (std::is_same<DT, float>::value) {
        dt = HIP_R_32F;
    } else if constexpr (std::is_same<DT, double>::value) {
        dt = HIP_R_64F;
    }
    assertTypes3(DT, half, float, double);

    CHECK_CUSPARSE(hipsparseCreateCoo(
        matDescP, this->numRows, this->numCols, this->numNonZero, this->rowIdxs,
        this->colIdxs, this->data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
        dt));
}

template <typename DT, typename MT>
hipsparseSpMMAlg_t SparseMatrixCOO<DT, MT>::getCusparseAlg() {
    return HIPSPARSE_SPMM_COO_ALG4;
}

template <typename DT, typename MT> SparseMatrixCOO<DT, MT> *SparseMatrixCOO<DT, MT>::copy2Device() {
    assert(this->onDevice == false);
    assert(this->data != nullptr);

    SparseMatrixCOO<DT, MT> *newMatrix = new SparseMatrixCOO<DT, MT>(
        this->numRows, this->numCols, this->numNonZero, true);

    cudaCheckError(hipMemcpy(newMatrix->rowIdxs, this->rowIdxs,
                              this->numNonZero * sizeof(DT),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->colIdxs, this->colIdxs,
                              this->numNonZero * sizeof(DT),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->data, this->data,
                              this->numNonZero * sizeof(DT),
                              hipMemcpyHostToDevice));
    return newMatrix;
}

template <typename DT, typename MT> bool SparseMatrixCOO<DT, MT>::allocateSpace(bool onDevice) {
    assert(this->data == nullptr);
    assert(this->rowIdxs == nullptr);
    assert(this->colIdxs == nullptr);
    if (onDevice) {
        cudaCheckError(hipMalloc(&this->data, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMalloc(
            &this->rowIdxs, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMalloc(
            &this->colIdxs, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMemset(this->data, 0, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMemset(
            this->rowIdxs, 0, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMemset(
            this->colIdxs, 0, this->numNonZero * sizeof(DT)));
    } else {
        cudaCheckError(
            hipHostMalloc(&this->data, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipHostMalloc(
            &this->rowIdxs, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipHostMalloc(
            &this->colIdxs, this->numNonZero * sizeof(DT)));
        std::memset(this->data, 0, this->numNonZero * sizeof(DT));
        std::memset(this->rowIdxs, 0,
                    this->numNonZero * sizeof(DT));
        std::memset(this->colIdxs, 0,
                    this->numNonZero * sizeof(DT));
    }

    return true;
}

template <typename DT, typename MT> DenseMatrix<DT, MT> *SparseMatrixCOO<DT, MT>::toDense() {
    assert(!this->onDevice);

    using mt = MT;

    DenseMatrix<DT, MT> *dm =
        new DenseMatrix<DT, MT>(this->numRows, this->numCols, false);

    for (size_t i = 0; i < this->numNonZero; i++) {
        mt r = this->rowIdxs[i];
        mt c = this->colIdxs[i];
        dm->data[r * dm->numCols + c] = this->data[i];
    }

    return dm;
}

template <typename DT, typename MT>
std::ostream &operator<<(std::ostream &out, SparseMatrixCOO<DT, MT> &m) {
    out << m.numRows << ' ' << m.numCols << ' ' << m.numNonZero << std::endl;
    for (size_t i = 0; i < m->numNonZero; i++) {
        std::cout << m.rowIdxs[i] << ' ' << m.colIdxs[i] << ' ' << m.data[i]
                  << std::endl;
    }

    return out;
}

template class SparseMatrixCOO<float, uint32_t>;
template class SparseMatrixCOO<double, uint32_t>;
} // namespace cuspmm