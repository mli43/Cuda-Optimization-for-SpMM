#include "commons.hpp"
#include "formats/sparse_csr.hpp"

namespace cuspmm {
template <typename DT, typename MT>
SparseMatrixCSR<DT, MT>::SparseMatrixCSR() : SparseMatrix<DT, MT>() {
    this->rowPtrs = nullptr;
    this->colIdxs = nullptr;
}

template <typename DT, typename MT>
SparseMatrixCSR<DT, MT>::SparseMatrixCSR(std::string filePath)
    : rowPtrs(nullptr), colIdxs(nullptr) {
    this->onDevice = false;

    std::ifstream inputFile(filePath);
    std::string line;

    if (!inputFile.is_open()) {
        std::cerr << "File " << filePath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    inputFile >> this->numRows >> this->numCols >> this->numNonZero;
    std::getline(inputFile, line); // Discard the line

    this->allocateSpace(false);

    // Read row ptrs
    std::getline(inputFile, line);
    std::istringstream iss(line);
    for (int i = 0; i <= this->numRows; i++) {
        iss >> this->rowPtrs[i];
    }

    // Read column index
    std::getline(inputFile, line);
    iss.str(line);
    iss.clear();
    for (int i = 0; i <= this->numNonZero; i++) {
        iss >> this->colIdxs[i];
    }

    // Read data
    std::getline(inputFile, line);
    iss.str(line);
    iss.clear();
    for (int i = 0; i < this->numNonZero; i++) {
        iss >> this->data[i];
    }
}

template <typename DT, typename MT>
SparseMatrixCSR<DT, MT>::SparseMatrixCSR(MT numRows,
                                    MT numCols,
                                    MT numNonZero,
                                    bool onDevice)
    : rowPtrs(nullptr), colIdxs(nullptr) {
    this->numRows = numRows;
    this->numCols = numCols;
    this->numNonZero = numNonZero;
    this->onDevice = onDevice;
    this->allocateSpace(onDevice);
}

template <typename DT, typename MT> SparseMatrixCSR<DT, MT>::~SparseMatrixCSR() {
    if (this->rowPtrs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->rowPtrs));
        } else {
            cudaCheckError(hipHostFree(this->rowPtrs));
        }
    }

    if (this->colIdxs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->colIdxs));
        } else {
            cudaCheckError(hipHostFree(this->colIdxs));
        }
    }

    if (this->data != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->data));
        } else {
            cudaCheckError(hipHostFree(this->data));
        }
    }
}

template <typename DT, typename MT>
void SparseMatrixCSR<DT, MT>::setCusparseSpMatDesc(hipsparseSpMatDescr_t *matDescP) {
    hipDataType dt;
    if constexpr (std::is_same<DT, half>::value) {
        dt = HIP_R_16F;
    } else if constexpr (std::is_same<DT, float>::value) {
        dt = HIP_R_32F;
    } else if constexpr (std::is_same<DT, double>::value) {
        dt = HIP_R_64F;
    }
    assertTypes3(DT, half, float, double);

    CHECK_CUSPARSE(hipsparseCreateCsr(matDescP, this->numRows, this->numCols, this->numNonZero, this->rowPtrs,
                      this->colIdxs, this->data, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, dt));
}


template <typename DT, typename MT> SparseMatrixCSR<DT, MT> *SparseMatrixCSR<DT, MT>::copy2Device() {
    assert(this->onDevice == false);
    assert(this->data != nullptr);

    SparseMatrixCSR<DT, MT> *newMatrix = new SparseMatrixCSR<DT, MT>(
        this->numRows, this->numCols, this->numNonZero, true);

    cudaCheckError(
        hipMemcpy(newMatrix->rowPtrs, this->rowPtrs,
                   (this->numRows + 1) * sizeof(DT),
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->colIdxs, this->colIdxs,
                              this->numNonZero * sizeof(DT),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->data, this->data,
                              this->numNonZero * sizeof(DT),
                              hipMemcpyHostToDevice));
    return newMatrix;
}

template <typename DT, typename MT> bool SparseMatrixCSR<DT, MT>::allocateSpace(bool onDevice) {
    assert(this->data == nullptr);
    if (onDevice) {
        cudaCheckError(hipMalloc(&this->data, this->numNonZero * sizeof(DT)));
        cudaCheckError(
            hipMalloc(&this->rowPtrs,
                       (this->numRows + 1) * sizeof(DT)));
        cudaCheckError(hipMalloc(
            &this->colIdxs, this->numNonZero * sizeof(DT)));
        cudaCheckError(hipMemset(this->data, 0, this->numNonZero * sizeof(DT)));
        cudaCheckError(
            hipMemset(this->rowPtrs, 0,
                       (this->numRows + 1) * sizeof(DT)));
        cudaCheckError(hipMemset(
            this->colIdxs, 0, this->numNonZero * sizeof(DT)));
    } else {
        cudaCheckError(
            hipHostMalloc(&this->data, this->numNonZero * sizeof(DT)));
        cudaCheckError(
            hipHostMalloc(&this->rowPtrs,
                           (this->numRows + 1) * sizeof(DT)));
        cudaCheckError(hipHostMalloc(
            &this->colIdxs, this->numNonZero * sizeof(DT)));
        std::memset(this->data, 0, this->numNonZero * sizeof(DT));
        std::memset(this->rowPtrs, 0,
                    (this->numRows + 1) * sizeof(DT));
        std::memset(this->colIdxs, 0,
                    this->numNonZero * sizeof(DT));
    }

    return true;
}

template <typename DT, typename MT> DenseMatrix<DT, MT> *SparseMatrixCSR<DT, MT>::toDense() {
    assert(!this->onDevice);

    using mt = MT;

    DenseMatrix<DT, MT> *dm =
        new DenseMatrix<DT, MT>(this->numRows, this->numCols, false);

    for (mt r = 0; r < this->numRows; r++) {
        mt row_start = this->rowPtrs[r];
        mt row_end = this->rowPtrs[r + 1];
        for (mt idx = row_start; idx < row_end; idx++) {
            mt c = this->colIdxs[idx];
            dm->data[r * dm->numCols + c] = this->data[idx];
        }
    }
    return dm;
}

template <typename DT, typename MT>
hipsparseSpMMAlg_t SparseMatrixCSR<DT, MT>::getCusparseAlg() {
    return HIPSPARSE_SPMM_CSR_ALG2;
}

template <typename DT, typename MT>
std::ostream &operator<<(std::ostream &out, SparseMatrixCSR<DT, MT> &m) {
    out << m.numRows << ' ' << m.numCols << ' ' << m.numNonZero << std::endl;
    for (size_t i = 0; i < m.numRows; i++) {
        out << m.rowPtrs[i] << ' ';
    }
    out << std::endl;

    for (size_t i = 0; i < m.numNonZero; i++) {
        out << m.colIdxs[i] << ' ';
    }
    out << std::endl;

    for (size_t i = 0; i < m.numNonZero; i++) {
        out << m.data[i] << ' ';
    }
    out << std::endl;

    return out;
}

template class SparseMatrixCSR<float, uint32_t>;
template class SparseMatrixCSR<double, uint32_t>;

} // namespace cuspmm