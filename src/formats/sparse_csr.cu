#include "commons.hpp"
#include "formats/sparse_csr.hpp"

namespace cuspmm {
template <typename T>
SparseMatrixCSR<T>::SparseMatrixCSR() : SparseMatrix<T>() {
    this->rowPtrs = nullptr;
    this->colIdxs = nullptr;
}

template <typename T>
SparseMatrixCSR<T>::SparseMatrixCSR(std::string filePath)
    : rowPtrs(nullptr), colIdxs(nullptr) {
    this->onDevice = false;

    std::ifstream inputFile(filePath);
    std::string line;

    if (!inputFile.is_open()) {
        std::cerr << "File " << filePath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    inputFile >> this->numRows >> this->numCols >> this->numNonZero;
    std::getline(inputFile, line); // Discard the line

    this->allocateSpace(false);

    // Read row ptrs
    std::getline(inputFile, line);
    std::istringstream iss(line);
    for (int i = 0; i <= this->numRows; i++) {
        iss >> this->rowPtrs[i];
    }

    // Read column index
    std::getline(inputFile, line);
    iss.str(line);
    iss.clear();
    for (int i = 0; i <= this->numNonZero; i++) {
        iss >> this->colIdxs[i];
    }

    // Read data
    std::getline(inputFile, line);
    iss.str(line);
    iss.clear();
    for (int i = 0; i < this->numNonZero; i++) {
        iss >> this->data[i];
    }
}

template <typename T>
SparseMatrixCSR<T>::SparseMatrixCSR(Matrix::metadataType numRows,
                                    Matrix::metadataType numCols,
                                    Matrix::metadataType numNonZero,
                                    bool onDevice)
    : rowPtrs(nullptr), colIdxs(nullptr) {
    this->numRows = numRows;
    this->numCols = numCols;
    this->numNonZero = numNonZero;
    this->onDevice = onDevice;
    this->allocateSpace(onDevice);
}

template <typename T> SparseMatrixCSR<T>::~SparseMatrixCSR() {
    if (this->rowPtrs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->rowPtrs));
        } else {
            cudaCheckError(hipHostFree(this->rowPtrs));
        }
    }

    if (this->colIdxs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->colIdxs));
        } else {
            cudaCheckError(hipHostFree(this->colIdxs));
        }
    }

    if (this->data != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->data));
        } else {
            cudaCheckError(hipHostFree(this->data));
        }
    }
}

template <typename T>
void SparseMatrixCSR<T>::setCusparseSpMatDesc(hipsparseSpMatDescr_t *matDescP) {
    hipDataType dt;
    if constexpr (std::is_same<T, half>::value) {
        dt = HIP_R_16F;
    } else if constexpr (std::is_same<T, float>::value) {
        dt = HIP_R_32F;
    } else if constexpr (std::is_same<T, double>::value) {
        dt = HIP_R_64F;
    }
    assertTypes3(T, half, float, double);

    CHECK_CUSPARSE(hipsparseCreateCsr(matDescP, this->numRows, this->numCols, this->numNonZero, this->rowPtrs,
                      this->colIdxs, this->data, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, dt));
}


template <typename T> SparseMatrixCSR<T> *SparseMatrixCSR<T>::copy2Device() {
    assert(this->onDevice == false);
    assert(this->data != nullptr);

    SparseMatrixCSR<T> *newMatrix = new SparseMatrixCSR<T>(
        this->numRows, this->numCols, this->numNonZero, true);

    cudaCheckError(
        hipMemcpy(newMatrix->rowPtrs, this->rowPtrs,
                   (this->numRows + 1) * sizeof(Matrix::metadataType),
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->colIdxs, this->colIdxs,
                              this->numNonZero * sizeof(Matrix::metadataType),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->data, this->data,
                              this->numNonZero * sizeof(T),
                              hipMemcpyHostToDevice));
    return newMatrix;
}

template <typename T> bool SparseMatrixCSR<T>::allocateSpace(bool onDevice) {
    assert(this->data == nullptr);
    if (onDevice) {
        cudaCheckError(hipMalloc(&this->data, this->numNonZero * sizeof(T)));
        cudaCheckError(
            hipMalloc(&this->rowPtrs,
                       (this->numRows + 1) * sizeof(Matrix::metadataType)));
        cudaCheckError(hipMalloc(
            &this->colIdxs, this->numNonZero * sizeof(Matrix::metadataType)));
        cudaCheckError(hipMemset(this->data, 0, this->numNonZero * sizeof(T)));
        cudaCheckError(
            hipMemset(this->rowPtrs, 0,
                       (this->numRows + 1) * sizeof(Matrix::metadataType)));
        cudaCheckError(hipMemset(
            this->colIdxs, 0, this->numNonZero * sizeof(Matrix::metadataType)));
    } else {
        cudaCheckError(
            hipHostMalloc(&this->data, this->numNonZero * sizeof(T)));
        cudaCheckError(
            hipHostMalloc(&this->rowPtrs,
                           (this->numRows + 1) * sizeof(Matrix::metadataType)));
        cudaCheckError(hipHostMalloc(
            &this->colIdxs, this->numNonZero * sizeof(Matrix::metadataType)));
        std::memset(this->data, 0, this->numNonZero * sizeof(T));
        std::memset(this->rowPtrs, 0,
                    (this->numRows + 1) * sizeof(Matrix::metadataType));
        std::memset(this->colIdxs, 0,
                    this->numNonZero * sizeof(Matrix::metadataType));
    }

    return true;
}

template <typename T> DenseMatrix<T> *SparseMatrixCSR<T>::toDense() {
    assert(!this->onDevice);

    using mt = Matrix::metadataType;

    DenseMatrix<T> *dm =
        new DenseMatrix<T>(this->numRows, this->numCols, false);

    for (mt r = 0; r < this->numRows; r++) {
        mt row_start = this->rowPtrs[r];
        mt row_end = this->rowPtrs[r + 1];
        for (mt idx = row_start; idx < row_end; idx++) {
            mt c = this->colIdxs[idx];
            dm->data[r * dm->numCols + c] = this->data[idx];
        }
    }
    return dm;
}

template <typename T>
hipsparseSpMMAlg_t SparseMatrixCSR<T>::getCusparseAlg() {
    return HIPSPARSE_SPMM_CSR_ALG2;
}

template <typename T>
std::ostream &operator<<(std::ostream &out, SparseMatrixCSR<T> &m) {
    out << m.numRows << ' ' << m.numCols << ' ' << m.numNonZero << std::endl;
    for (size_t i = 0; i < m.numRows; i++) {
        out << m.rowPtrs[i] << ' ';
    }
    out << std::endl;

    for (size_t i = 0; i < m.numNonZero; i++) {
        out << m.colIdxs[i] << ' ';
    }
    out << std::endl;

    for (size_t i = 0; i < m.numNonZero; i++) {
        out << m.data[i] << ' ';
    }
    out << std::endl;

    return out;
}

template class SparseMatrixCSR<float>;
template class SparseMatrixCSR<double>;

} // namespace cuspmm