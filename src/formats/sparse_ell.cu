#include "hip/hip_runtime.h"
#include "formats/sparse_ell.hpp"

namespace cuspmm {

template <typename T>
SparseMatrixELL<T>::SparseMatrixELL() : SparseMatrix<T>() {
    this->colIdxs = nullptr;
    this->maxRowNnz = 0;
}

template <typename T>
SparseMatrixELL<T>::SparseMatrixELL(std::string colindPath,
                                    std::string valuesPath) {
    this->colIdxs = nullptr;
    this->onDevice = false;

    std::ifstream colindFile(colindPath);
    std::string line_colind;

    std::ifstream valuesFile(valuesPath);
    std::string line_values;

    if (!colindFile.is_open()) {
        std::cerr << "File " << colindPath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    if (!valuesFile.is_open()) {
        std::cerr << "File " << valuesPath << "doesn't exist!" << std::endl;
        throw std::runtime_error(NULL);
    }

    colindFile >> this->numRows >> this->numCols >> this->numNonZero >>
        this->maxRowNnz;
    constexpr auto max_size = std::numeric_limits<std::streamsize>::max();
    colindFile.ignore(max_size, '\n');

    this->allocateSpace(false);

    // Read col indexes
    for (size_t row = 0; row < this->numRows; row++) {
        for (size_t i = 0; i < this->maxRowNnz; i++) {
            colindFile >> this->colIdxs[row * this->maxRowNnz + i];
        }
    }

    // Read values
    for (size_t row = 0; row < this->numRows; row++) {
        for (size_t i = 0; i < this->maxRowNnz; i++) {
            valuesFile >> this->data[row * this->maxRowNnz + i];
        }
    }

    colindFile.close();
    valuesFile.close();
}

template <typename T>
SparseMatrixELL<T>::SparseMatrixELL(Matrix::metadataType numRows,
                                    Matrix::metadataType numCols,
                                    Matrix::metadataType numNonZero,
                                    Matrix::metadataType maxRowNnz,
                                    bool onDevice) {
    this->numRows = numRows;
    this->numCols = numCols;
    this->numNonZero = numNonZero;
    this->onDevice = onDevice;
    this->maxRowNnz = maxRowNnz;
    this->colIdxs = nullptr;
    this->data = nullptr;
    this->allocateSpace(onDevice);
}

template <typename T> SparseMatrixELL<T>::~SparseMatrixELL() {
    if (this->colIdxs != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->colIdxs));
        } else {
            cudaCheckError(hipHostFree(this->colIdxs));
        }
    }

    if (this->data != nullptr) {
        if (this->onDevice) {
            cudaCheckError(hipFree(this->data));
        } else {
            cudaCheckError(hipHostFree(this->data));
        }
    }
}

template <typename T> bool SparseMatrixELL<T>::allocateSpace(bool onDevice) {
    assert(this->data == nullptr);
    assert(this->colIdxs == nullptr);
    if (onDevice) {

        cudaCheckError(hipMalloc(&this->data,
                                  this->numRows * this->maxRowNnz * sizeof(T)));
        cudaCheckError(
            hipMalloc(&this->colIdxs, this->numRows * this->maxRowNnz *
                                           sizeof(Matrix::metadataType)));
        cudaCheckError(hipMemset(this->data, 0,
                                  this->numRows * this->maxRowNnz * sizeof(T)));
        cudaCheckError(hipMemset(this->colIdxs, 0,
                                  this->numRows * this->maxRowNnz *
                                      sizeof(Matrix::metadataType)));
    } else {
        cudaCheckError(hipHostMalloc(
            &this->data, this->numRows * this->maxRowNnz * sizeof(T)));
        cudaCheckError(
            hipHostMalloc(&this->colIdxs, this->numRows * this->maxRowNnz *
                                               sizeof(Matrix::metadataType)));
        std::memset(this->data, 0, this->numRows * this->maxRowNnz * sizeof(T));
        std::memset(this->colIdxs, 0,
                    this->numRows * this->maxRowNnz *
                        sizeof(Matrix::metadataType));
    }

    return true;
}

template <typename T> SparseMatrixELL<T> *SparseMatrixELL<T>::copy2Device() {
    assert(this->onDevice == false);
    assert(this->data != nullptr);

    SparseMatrixELL<T> *newMatrix = new SparseMatrixELL<T>(
        this->numRows, this->numCols, this->numNonZero, this->maxRowNnz, true);

    cudaCheckError(hipMemcpy(newMatrix->colIdxs, this->colIdxs,
                              this->numRows * this->maxRowNnz *
                                  sizeof(Matrix::metadataType),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(newMatrix->data, this->data,
                              this->numRows * this->maxRowNnz * sizeof(T),
                              hipMemcpyHostToDevice));

    return newMatrix;
}

template <typename T> DenseMatrix<T> *SparseMatrixELL<T>::toDense() {
    assert(!this->onDevice);

    using mt = Matrix::metadataType;

    DenseMatrix<T> *dm =
        new DenseMatrix<T>(this->numRows, this->numCols, false);

    for (size_t row = 0; row < this->numRows; row++) {
        size_t base = row * this->maxRowNnz;
        for (size_t colind = 0; colind < this->maxRowNnz; colind++) {
            int col = this->colIdxs[base + colind];
            if (col >= 0) {
                dm->data[row * dm->numCols + col] = this->data[base + colind];
            }
        }
    }

    return dm;
}

template class SparseMatrixELL<float>;
template class SparseMatrixELL<double>;

} // namespace cuspmm