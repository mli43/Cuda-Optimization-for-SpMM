#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "commons.hpp"
#include "formats/sparse_bsr.hpp"
#include <cstdint>

namespace cuspmm {

template <typename DT, typename MT, typename AccT>
__global__ void spmmBSRK1(MT aNumRows, MT aNumCols, MT aBlockRowSize, MT aBlockColSize,
                                    MT aNumBlocks,
                                    MT *aBlockRowPtrs, MT *aBlockColIdxs, DT* aData, 
                                    MT bNumRows, MT bNumCols, DT* bData,
                                    DT* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    // Every thread block is responsible for one `a` block row
    unsigned int blockRowIdx = blockIdx.x;
    unsigned int inBlockRow = threadIdx.x;
    unsigned int inBlockCol = threadIdx.y;
    AccT accumulator = 0.f;

    unsigned int blockRowStartIdx = aBlockRowPtrs[blockRowIdx];
    unsigned int blockRowEndIdx = aBlockRowPtrs[blockRowIdx + 1];

    const unsigned int aDenseRowBase = blockRowIdx * aBlockRowSize;
    for (unsigned aBlockIdx = blockRowStartIdx; aBlockIdx < blockRowEndIdx; aBlockIdx++) {
        unsigned int aBlockCol = aBlockColIdxs[aBlockIdx];
        const unsigned int aDenseColBase = aBlockCol * aBlockColSize;
        DT* aBlockData = aData + aBlockIdx * aBlockRowSize * aBlockColSize;

        DT aDataElement = aBlockData[RowMjIdx(inBlockRow, inBlockCol, aBlockColSize)];
        unsigned int ar = aDenseRowBase + inBlockRow;
        unsigned int ac = aDenseColBase + inBlockCol;

        for (unsigned bc = 0; bc < bNumCols; bc++) {
            // ! This can be improved. Accumulate locally
            atomicAdd(&cData[RowMjIdx(ar, bc, bNumCols)], aDataElement * bData[RowMjIdx(ac, bc, bNumCols)]);
        }
    }
}

template <typename DT, typename MT, typename AccT>
DenseMatrix<DT, MT>* spmmBSRWrapper1(SparseMatrixBSR<DT, MT>* a, DenseMatrix<DT, MT>* b, DenseMatrix<DT, MT>* c) {
    assert(a->onDevice && b->onDevice);
    size_t rows = a->numCols, cols = b->numCols;

    if (b->ordering == ORDERING::COL_MAJOR) {
        b->toOrdering(ORDERING::ROW_MAJOR);
    }

    // (x, y)
    dim3 block(a->blockRowSize, a->blockColSize);
    dim3 grid(a->numBlockRows);

    auto t1 = std::chrono::high_resolution_clock::now();
    spmmBSRK1<DT, MT, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->blockRowSize, a->blockColSize,
        a->numBlockRows, a->blockRowPtrs, a->blockColIdxs,
        a->data, b->numRows, b->numCols, b->data, 
        c->data
    );

    hipDeviceSynchronize();

    auto t2 = std::chrono::high_resolution_clock::now();
    printf("%s with shape block(z=%d,y=%d,x=%d) grid(z=%d,y=%d,x=%d): %ld ns\n", __func__,
            block.z, block.y, block.x, grid.z, grid.y, grid.x, std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count());
    
    return c;
}

// instantiations
template DenseMatrix<float, uint32_t>* spmmBSRWrapper1<float, uint32_t, double>(SparseMatrixBSR<float, uint32_t>* a, DenseMatrix<float, uint32_t>* b, DenseMatrix<float, uint32_t>* c);


} // namespace cuspmm
