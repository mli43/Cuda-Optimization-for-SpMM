#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "commons.hpp"
#include "formats/sparse_coo.hpp"
#include <cstdint>

namespace cuspmm {

template <typename DT, typename MT, typename AccT>
__global__ void spmmCOOK1(MT aNumRows, MT aNumCols, MT aNumNonZero,
                                    MT *rowIdxs, MT *colIdxs, DT* aData, 
                                    MT bNumRows, MT bNumCols, DT* bData,
                                    DT* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < aNumNonZero) {
        int row = rowIdxs[idx];
        int col = colIdxs[idx];
        float value = aData[idx];

        for (int j = 0; j < bNumCols; j++) {
            atomicAdd(&cData[row * bNumCols + j], value * bData[col * bNumCols + j]);
        }
    }
}

template <typename DT, typename MT, typename AccT>
__global__ void spmmCOOK2(MT aNumRows, MT aNumCols, MT aNumNonZero,
                                    MT *rowIdxs, MT *colIdxs, DT* aData, 
                                    MT bNumRows, MT bNumCols, DT* bData,
                                    DT* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < aNumNonZero) {
        int row = rowIdxs[idx];
        int col = colIdxs[idx];
        float value = aData[idx];

        for (int j = 0; j < bNumCols; j++) {
            atomicAdd(&cData[row * bNumCols + j], value * bData[col * bNumCols + j]);
        }
    }
}

template <typename DT, typename MT, typename AccT>
DenseMatrix<DT, MT>* spmmCOOWrapper1(SparseMatrixCOO<DT, MT>* a, DenseMatrix<DT, MT>* b, DenseMatrix<DT, MT>* ref) {

    if (b->ordering == ORDERING::COL_MAJOR) {
        b->toOrdering(ORDERING::ROW_MAJOR);
    }

    const int kernelNum = 1;
    assert(a->onDevice && b->onDevice);

    // 1. Prologue
    auto t1 = std::chrono::high_resolution_clock::now();
    size_t rows = a->numRows, cols = b->numCols;
    const size_t BLOCKSIZE = 1024;
    const size_t numNonZero = a->numNonZero;

    auto* c = new DenseMatrix<DT, MT>(rows, cols, true, ORDERING::ROW_MAJOR);

    dim3 block(BLOCKSIZE);
    dim3 grid((numNonZero + BLOCKSIZE - 1) / BLOCKSIZE);

    auto t2 = std::chrono::high_resolution_clock::now();
    spmmCOOK1<DT, MT, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->numNonZero, a->rowIdxs, a->colIdxs, a->data,
        b->numRows, b->numCols, b->data, 
        c->data
    );
    hipDeviceSynchronize();

    // 3. Epilogue
    auto t3 = std::chrono::high_resolution_clock::now();
    // printf("%s with shape block(z=%d,y=%d,x=%d) grid(z=%d,y=%d,x=%d): %ld ns\n", __func__,
    //         block.z, block.y, block.x, grid.z, grid.y, grid.x, std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count());
    
    auto res = c->copy2Host();
    auto t4 = std::chrono::high_resolution_clock::now();

    auto pro = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
    auto kernel = std::chrono::duration_cast<std::chrono::microseconds>(t3 - t2).count();
    auto epi = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();

    // Check correctness
    torch::Tensor refTorch = toTorch<DT, DenseMatrix<DT, MT>>(ref);
    torch::Tensor cTorch = toTorch<DT, DenseMatrix<DT, MT>>(res);
    bool correct = torch::allclose(cTorch, refTorch, REL_TOL, ABS_TOL);

    reportTime(testcase, a->numRows, a->numCols, a->numNonZero, std::string("COO"), 
        b->ordering, kernelNum, (double)(pro) / 1000, (double)(kernel) / 1000, (double)(epi) / 1000, correct);
    
    return c;
}

// Instantiation
template DenseMatrix<float, uint32_t>* spmmCOOWrapper1<float, uint32_t, double>(SparseMatrixCOO<float, uint32_t>* a, DenseMatrix<float, uint32_t>* b, DenseMatrix<float, uint32_t>* c) __attribute__((used));

} // namespace cuspmm
