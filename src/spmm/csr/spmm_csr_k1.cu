#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "commons.hpp"
#include "formats/matrix.hpp"
#include "formats/sparse_csr.hpp"
#include <cstdint>

namespace cuspmm {

template <typename DT, typename MT, typename AccT>
__global__ void spmmCSRK1(MT aNumRows, MT aNumCols, MT aNumNonZero,
                                    MT *rowPtrs, MT *colIdxs, DT* aData, 
                                    MT bNumRows, MT bNumCols, DT* bData,
                                    DT* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;

    if (c < bNumCols and r < aNumRows) {
        AccT acc = .0f;
        unsigned int row_start = rowPtrs[r];
        unsigned int row_end = rowPtrs[r + 1];

        for (unsigned int i = row_start; i < row_end; i++) {
            unsigned int c_idx = colIdxs[i];
            DT aValue = aData[i];
            acc += aValue * bData[c_idx * bNumCols + c];
        }
        cData[r * bNumCols + c] = acc;
    }
}

template <typename DT, typename MT, typename AccT>
DenseMatrix<DT, MT>* spmmCSRWrapper1(SparseMatrixCSR<DT, MT>* a, DenseMatrix<DT, MT>* b, DenseMatrix<DT, MT>* c) {
    size_t rows = a->numCols, cols = b->numCols;

    if (b->ordering == ORDERING::COL_MAJOR) {
        b->toOrdering(ORDERING::ROW_MAJOR);
    }

    const size_t BLOCKSIZE = 32;

    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((cols + BLOCKSIZE - 1) / BLOCKSIZE, (rows + BLOCKSIZE - 1) / BLOCKSIZE);

    assert(a->onDevice && b->onDevice);

    auto t1 = std::chrono::high_resolution_clock::now();
    spmmCSRK1<DT, MT, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->numNonZero, a->rowPtrs, a->colIdxs, a->data,
        b->numRows, b->numCols, b->data, 
        c->data
    );
    hipDeviceSynchronize();

    auto t2 = std::chrono::high_resolution_clock::now();
    printf("%s with shape block(z=%d,y=%d,x=%d) grid(z=%d,y=%d,x=%d): %ld ns\n", __func__,
            block.z, block.y, block.x, grid.z, grid.y, grid.x, std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count());
    
    return c;
}

template DenseMatrix<float, uint32_t>* spmmCSRWrapper1<float, uint32_t, double>(SparseMatrixCSR<float, uint32_t>* a, DenseMatrix<float, uint32_t>* b, DenseMatrix<float, uint32_t>* c);

} // namespace cuspmm
