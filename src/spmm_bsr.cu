#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "torch/torch.h"
#include "spmm_bsr.hpp"
#include <cassert>
#include <cstring>
#include <iostream>

namespace cuspmm {

template <typename T, typename MT, typename AccT>
__global__ void spmmBSRK1(MT aNumRows, MT aNumCols, MT aBlockSize,
                                    MT *aNumBlocks,
                                    MT *aBlockRowPtrs, MT *aBlockColIdxs, T* aData, 
                                    MT bNumRows, MT bNumCols, T* bData,
                                    T* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    // Every thread is responsible for one `a` block
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    AccT acc = .0f;
    unsigned int aDenseRowStart = aBlockRowPtrs[];
    unsigned int row_end = rowPtrs[r + 1];

    for (unsigned int i = row_start; i < row_end; i++) {
        unsigned int c_idx = colIdxs[i];
        T aValue = aData[i];
        acc += aValue * bData[c_idx * bNumCols + c];
    }
    cData[r * bNumCols + c] = acc;
}

template <typename T, typename AccT>
DenseMatrix<T>* spmmBsrDevice(SparseMatrixBSR<T>* a, DenseMatrix<T>* b) {
    size_t rows = a->numCols, cols = b->numCols;

    const size_t BLOCKSIZE = 32;

    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((cols + BLOCKSIZE - 1) / BLOCKSIZE, (rows + BLOCKSIZE - 1) / BLOCKSIZE);

    if (!a->onDevice || !b->onDevice) {
        std::cerr << "Device incorrect!" << std::endl; 
        return nullptr;
    }

    DenseMatrix<T>* c = new DenseMatrix<T>(a->numRows, b->numCols, true);

    spmmCSRK1<T, typename SparseMatrixCSR<T>::metadataType, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->numNonZero, a->rowPtrs, a->colIdxs, a->data,
        b->numRows, b->numCols, b->data, 
        c->data
    );

    return c;
}

template <typename T>
void runEngineCSR(SparseMatrixBSR<T> *a, DenseMatrix<T>* b, float abs_tol, double rel_tol) {

    // 1. Move to device
    SparseMatrixBSR<T>* da = a->copy2Device();
    DenseMatrix<T>* db = b->copy2Device();

    // 2. Launch kernel
    auto cRes = spmmBsrDevice<T, double>(da, db);
    auto cResCpu = cRes->copy2Host();
    cResCpu->save2File("bsr_cuda.res");

    // 3. Check result
    auto cResSeq = spmmBsrCpu<T, double>(a, b);
    cResSeq->save2File("bsr_cpu.res");

    auto denseA = a->toDense();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor taDevice = torch::from_blob(denseA->data, {denseA->numRows, denseA->numCols}, options).clone().cuda();
    torch::Tensor tbDevice = torch::from_blob(b->data, {b->numRows, b->numCols}, options).clone().cuda();
    torch::Tensor tcCpu = torch::from_blob(cResCpu->data, {cResCpu->numRows, cResCpu->numCols}, options).clone();
    torch::Tensor cResTorch = torch::matmul(taDevice, tbDevice).cpu();
    std::cout << "csr allclose: " << torch::allclose(tcCpu, cResTorch, rel_tol, abs_tol) << std::endl;

    auto denseTorch = new DenseMatrix<T>(cResCpu->numRows, cResCpu->numCols, false);
    std::memcpy(denseTorch->data, cResTorch.data_ptr<float>(), denseTorch->numRows * denseTorch->numCols * sizeof(float));
    denseTorch->save2File("bsr_torch.res");
}

template void runEngineCSR<float>(SparseMatrixBSR<float> *a, DenseMatrix<float>* b, float abs_tol, double rel_tol);

} // namespace cuspmm