#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "spmm_coo.hpp"
#include "torch/torch.h"
#include "engine.hpp"
#include "spmm_cusparse.hpp"
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstring>
#include <iostream>

namespace cuspmm {

template <typename T, typename MT, typename AccT>
__global__ void spmmCOOK1(MT aNumRows, MT aNumCols, MT aNumNonZero,
                                    MT *rowIdxs, MT *colIdxs, T* aData, 
                                    MT bNumRows, MT bNumCols, T* bData,
                                    T* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < aNumNonZero) {
        int row = rowIdxs[idx];
        int col = colIdxs[idx];
        float value = aData[idx];

        for (int j = 0; j < bNumCols; j++) {
            atomicAdd(&cData[row * bNumCols + j], value * bData[col * bNumCols + j]);
        }
    }
}

template <typename T, typename MT, typename AccT>
__global__ void spmmCOOK2(MT aNumRows, MT aNumCols, MT aNumNonZero,
                                    MT *rowIdxs, MT *colIdxs, T* aData, 
                                    MT bNumRows, MT bNumCols, T* bData,
                                    T* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < aNumNonZero) {
        int row = rowIdxs[idx];
        int col = colIdxs[idx];
        float value = aData[idx];

        for (int j = 0; j < bNumCols; j++) {
            atomicAdd(&cData[row * bNumCols + j], value * bData[col * bNumCols + j]);
        }
    }
}

template <typename T, typename AccT>
DenseMatrix<T>* spmmCooDevice(SparseMatrixCOO<T>* a, DenseMatrix<T>* b) {
    const size_t numNonZero = a->numNonZero;

    const size_t BLOCKSIZE = 1024;

    dim3 block(BLOCKSIZE);
    dim3 grid((numNonZero + BLOCKSIZE - 1) / BLOCKSIZE);

    if (!a->onDevice || !b->onDevice) {
        std::cerr << "Device incorrect!" << std::endl; 
        return nullptr;
    }

    DenseMatrix<T>* c = new DenseMatrix<T>(a->numRows, b->numCols, true);

    spmmCOOK1<T, typename SparseMatrixCOO<T>::metadataType, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->numNonZero, a->rowIdxs, a->colIdxs, a->data,
        b->numRows, b->numCols, b->data, 
        c->data
    );

    return c;
}

/**
 * @brief Cusparse spmm
 * 
 * @tparam T 
 * @param a COO format, must be on device!
 * @param b dense matrix b must COL_MAJOR and on device!
 * @return DenseMatrix<T>* Will be row-major
 */
template <typename T>
DenseMatrix<T>* spmmCOOCuSparse(SparseMatrixCOO<T>* a, DenseMatrix<T>* b) {
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    DenseMatrix<T>* c = new DenseMatrix<T>(a->numRows, b->numCols, true, ORDERING::ROW_MAJOR);

    if (b->ordering != ORDERING::COL_MAJOR) {
        b->toOrdering(ORDERING::COL_MAJOR);
    }

    auto t1 = std::chrono::high_resolution_clock::now();

    CHECK_CUSPARSE(hipsparseCreate(&handle));
    // FIXME: Only supports float right not!
    CHECK_CUSPARSE(hipsparseCreateCoo(&matA, a->numRows, a->numCols, a->numNonZero, a->rowIdxs,a->colIdxs, a->data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, b->numRows, b->numCols, b->numRows, b->data, HIP_R_32F, HIPSPARSE_ORDER_COL));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, a->numRows, b->numCols, b->numCols, c->data, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    float alpha = 1.0f, beta = 0.f;
    void* dBuffer = nullptr;
    size_t buffersize = 0;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, &buffersize));
    cudaCheckError(hipMalloc(&dBuffer, buffersize));
    auto t2 = std::chrono::high_resolution_clock::now();
    CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2, dBuffer));
    hipDeviceSynchronize();
    auto t3 = std::chrono::high_resolution_clock::now();

    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    cudaCheckError(hipFree(dBuffer));
    auto t4 = std::chrono::high_resolution_clock::now();

    auto prepTime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(t3 - t2);
    auto epilogueTime = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3);

    std::cout << "cusparse prep time (us):" << prepTime.count() << ','
              << "cusparse kernel time (us):" << kernelTime.count() << ','
              << "cusparse epilogue time (us):" << epilogueTime.count() << std::endl;
    
    return c;
}

template <typename T>
void runEngineCOO(SparseMatrixCOO<T> *a, DenseMatrix<T>* b, float abs_tol, double rel_tol) {
    auto start = std::chrono::high_resolution_clock::now();

    // 1. Move to device
    SparseMatrixCOO<T>* da = a->copy2Device();
    DenseMatrix<T>* db = b->copy2Device();
    auto copy_to_device_end = std::chrono::high_resolution_clock::now();

    // 2. Launch kernel
    auto cRes = spmmCooDevice<T, double>(da, db);
    auto kernel_end = std::chrono::high_resolution_clock::now();

    auto cResCpu = cRes->copy2Host();
    auto copy_to_host_end = std::chrono::high_resolution_clock::now();

    // 3. Check result
    auto cResSeq = spmmCooCpu<T, double>(a, b);
    auto seq_end = std::chrono::high_resolution_clock::now();

    // 4. Report time 
    auto copy2DeviceTime = std::chrono::duration_cast<std::chrono::microseconds>(copy_to_device_end - start);
    auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - copy_to_device_end);
    auto copy2HostTime = std::chrono::duration_cast<std::chrono::microseconds>(copy_to_host_end - kernel_end);
    auto parallelTime = std::chrono::duration_cast<std::chrono::microseconds>(copy_to_host_end - start);
    auto seqTime = std::chrono::duration_cast<std::chrono::microseconds>(seq_end - copy_to_host_end);

    std::cout << "copy2DeviceTime (us):" << copy2DeviceTime.count() << ','
              << "kernelTime (us):" << kernelTime.count() << ','
              << "copy2HostTime (us):" << copy2HostTime.count() << ','
              << "parallelTime (us):" << parallelTime.count() << ','
              << "seqTime (us):" << seqTime.count() << '\n';


    cResCpu->save2File("coo_cuda.res");
    cResSeq->save2File("coo_cpu.res");

    // cusparse test
    DenseMatrix<T>* bColMj = new DenseMatrix<T>(b, false);
    bColMj->toOrdering(ORDERING::COL_MAJOR);
    auto dbColMj = bColMj->copy2Device();
    auto cResCuSparse = cusparseTest<T>(da, dbColMj);
    auto cResCuSparseCpu = cResCuSparse->copy2Host();
    cResCuSparseCpu->save2File("coo_cusparse.res");

    auto denseA = a->toDense();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor taDevice = torch::from_blob(denseA->data, {denseA->numRows, denseA->numCols}, options).clone().cuda();
    torch::Tensor tbDevice = torch::from_blob(b->data, {b->numRows, b->numCols}, options).clone().cuda();
    torch::Tensor tcCpu = torch::from_blob(cResCpu->data, {cResCpu->numRows, cResCpu->numCols}, options).clone();
    torch::Tensor cResTorch = torch::matmul(taDevice, tbDevice).cpu();
    std::cout << "coo allclose: " << torch::allclose(tcCpu, cResTorch, rel_tol, abs_tol) << std::endl;

    auto denseTorch = new DenseMatrix<T>(cResCpu->numRows, cResCpu->numCols, false);
    std::memcpy(denseTorch->data, cResTorch.data_ptr<float>(), denseTorch->numRows * denseTorch->numCols * sizeof(float));
    denseTorch->save2File("coo_torch.res");
}

template void runEngineCOO<float>(SparseMatrixCOO<float> *a, DenseMatrix<float>* b, float abs_tol, double rel_tol);

} // namespace cuspmm
