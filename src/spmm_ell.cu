#include "hip/hip_runtime.h"
#include "cuda_utils.hpp"
#include "spmm_ell.hpp"
#include "torch/torch.h"
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <chrono>

namespace cuspmm {

template <typename T, typename MT, typename AccT>
__global__ void spmmELLK1(MT aNumRows, MT aNumCols, MT aNumNonZero, MT aMaxRowNnz,
                                    MT *colIdxs, T* aData, 
                                    MT bNumRows, MT bNumCols, T* bData,
                                    T* cData) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> C x N
    // C -> dense matrix -> C = A @ B -> R x N
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    size_t numValues = aNumRows * aMaxRowNnz;

    if (idx < numValues) {
        int row = idx / aMaxRowNnz;
        int col = colIdxs[idx];
        float value = aData[idx];
        

        if (col >= 0) {
            for (int j = 0; j < bNumCols; j++) {
                atomicAdd(&cData[row * bNumCols + j], value * bData[col * bNumCols + j]); 
            }
        }
    }
}

template <typename T, typename AccT>
DenseMatrix<T>* spmmEllDevice(SparseMatrixELL<T>* a, DenseMatrix<T>* b) {
    const size_t numValues = a->numRows * a->maxRowNnz;

    const size_t BLOCKSIZE = 1024;

    dim3 block(BLOCKSIZE);
    dim3 grid((numValues + BLOCKSIZE - 1) / BLOCKSIZE);

    if (!a->onDevice || !b->onDevice) {
        std::cerr << "Device incorrect!" << std::endl; 
        return nullptr;
    }

    DenseMatrix<T>* c = new DenseMatrix<T>(a->numRows, b->numCols, true);

    spmmELLK1<T, typename SparseMatrixELL<T>::metadataType, AccT><<<grid, block>>>(
        a->numRows, a->numCols, a->numNonZero, a->maxRowNnz, a->colIdxs, a->data,
        b->numRows, b->numCols, b->data, 
        c->data
    );

    return c;
}

template <typename T>
void runEngineELL(SparseMatrixELL<T> *a, DenseMatrix<T>* b, float abs_tol, double rel_tol) {
    auto start = std::chrono::high_resolution_clock::now();

    // 1. Move to device
    SparseMatrixELL<T>* da = a->copy2Device();
    DenseMatrix<T>* db = b->copy2Device();
    auto copy_to_device_end = std::chrono::high_resolution_clock::now();

    // 2. Launch kernel
    auto cRes = spmmEllDevice<T, double>(da, db);
    auto kernel_end = std::chrono::high_resolution_clock::now();

    auto cResCpu = cRes->copy2Host();
    auto copy_to_host_end = std::chrono::high_resolution_clock::now();

    // 3. Check result
    auto cResSeq = spmmEllCpu<T, double>(a, b);
    auto seq_end = std::chrono::high_resolution_clock::now();

    // 4. Report time 
    auto copy2DeviceTime = std::chrono::duration_cast<std::chrono::microseconds>(copy_to_device_end - start);
    auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - copy_to_device_end);
    auto copy2HostTime = std::chrono::duration_cast<std::chrono::microseconds>(copy_to_host_end - kernel_end);
    auto seqTime = std::chrono::duration_cast<std::chrono::microseconds>(seq_end - copy_to_host_end);

    std::cout << "copy2DeviceTime (us):" << copy2DeviceTime.count() << ','
              << "kernelTime (us):" << kernelTime.count() << ','
              << "copy2HostTime (us):" << copy2HostTime.count() << ','
              << "seqTime (us):" << seqTime.count() << '\n';

    cResCpu->save2File("ell_cuda.res");
    cResSeq->save2File("ell_cpu.res");

    auto denseA = a->toDense();
    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor taDevice = torch::from_blob(denseA->data, {denseA->numRows, denseA->numCols}, options).clone().cuda();
    torch::Tensor tbDevice = torch::from_blob(b->data, {b->numRows, b->numCols}, options).clone().cuda();
    torch::Tensor tcCpu = torch::from_blob(cResCpu->data, {cResCpu->numRows, cResCpu->numCols}, options).clone();
    torch::Tensor cResTorch = torch::matmul(taDevice, tbDevice).cpu();
    std::cout << "ell allclose: " << torch::allclose(tcCpu, cResTorch, rel_tol, abs_tol) << std::endl;

    auto denseTorch = new DenseMatrix<T>(cResCpu->numRows, cResCpu->numCols, false);
    std::memcpy(denseTorch->data, cResTorch.data_ptr<float>(), denseTorch->numRows * denseTorch->numCols * sizeof(float));
    denseTorch->save2File("ell_torch.res");
}

template void runEngineELL<float>(SparseMatrixELL<float> *a, DenseMatrix<float>* b, float abs_tol, double rel_tol);

} // namespace cuspmm
